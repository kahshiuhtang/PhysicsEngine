#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

__global__ void naive_MM(int M, int N, int K, float alpha, const float *A,
                            const float *B, float beta, float *C) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  // if statement is necessary to make things work under tile quantization
  if (x < M && y < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[x * K + i] * B[i * N + y];
    }
    // C = α*(A@B)+β*C
    C[x * N + y] = alpha * tmp + beta * C[x * N + y];
  }
}

void kernel(double *A, double *B, double *C, int arraySize) {

    // Initialize device pointers.
    double *d_A, *d_B, *d_C;

    // Allocate device memory.
    hipMalloc((void**) &d_A, arraySize * sizeof(double));
    hipMalloc((void**) &d_B, arraySize * sizeof(double));
    hipMalloc((void**) &d_C, arraySize * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / arraySize + 1, 1);

    // Launch CUDA kernel.

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);
}